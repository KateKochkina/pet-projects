#include "hip/hip_runtime.h"
%%writefile main1.cu
#include "hip/hip_runtime.h"
#include ""
#include <chrono>
#include <fstream>
#include <functional>
#include <iomanip>
#include <iostream>
#include <random>
#include <vector>

using namespace std;

int N = 10000;
int BS = 1024;
__device__ const double G = 6.67e-11;
__device__ const double EPS = 1e-15;


template<typename T>
ostream& operator<<(ostream& stream, const vector<T>& vec) {
    for (auto it = vec.begin(); it != vec.end(); ++it) {
        stream << " " << *it;
    }
    return stream;
}

__global__ void plus_product(double* result, double* lhs, double tau, double* rhs, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        result[idx] = lhs[idx] + tau * rhs[idx];
    }
}

__global__ void plus_equal_product(double* result, double tau, double* rhs, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        result[idx] += tau * rhs[idx];
    }
}

//__global__ void calc_k(double* result, double* pos_vel, double* masses, int n) {
//    int idx = blockIdx.x * blockDim.x + threadIdx.x;
//    if (idx < n) {
//        result[3 * idx + 0] = pos_vel[3 * n + 3 * idx + 0];
//        result[3 * idx + 1] = pos_vel[3 * n + 3 * idx + 1];
//        result[3 * idx + 2] = pos_vel[3 * n + 3 * idx + 2];
//        double a_x = 0.0, a_y = 0.0, a_z = 0.0;
//        double dist3, coef;
//        for (int j = 0; j < n; ++j) {
//            dist3 = pow(pow(pos_vel[3 * idx + 0] - pos_vel[3 * j + 0], 2) +
//                                pow(pos_vel[3 * idx + 1] - pos_vel[3 * j + 1], 2) +
//                                pow(pos_vel[3 * idx + 2] - pos_vel[3 * j + 2], 2),
//                        3.0 / 2);
//            coef = masses[j] / max(dist3, EPS);
//            a_x += coef * (pos_vel[3 * idx + 0] - pos_vel[3 * j + 0]);
//            a_y += coef * (pos_vel[3 * idx + 1] - pos_vel[3 * j + 1]);
//            a_z += coef * (pos_vel[3 * idx + 2] - pos_vel[3 * j + 2]);
//        }
//        result[3 * n + 3 * idx + 0] = -G * a_x;
//        result[3 * n + 3 * idx + 1] = -G * a_y;
//        result[3 * n + 3 * idx + 2] = -G * a_z;
//    }
//}

__device__ double sqr(double x) {
    return x * x;
}

__device__ double cube(double x) {
    return x * x * x;
}

__global__ void calc_k(double* result, double* pos_vel, double* masses, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        int n3 = 3 * n;
        int idx3 = 3 * idx;
        result[idx3] = pos_vel[n3 + idx3];
        result[idx3 + 1] = pos_vel[n3 + idx3 + 1];
        result[idx3 + 2] = pos_vel[n3 + idx3 + 2];

        /* copy all positions to shared memory */
        extern __shared__ double pos[];
        pos[idx3] = pos_vel[idx3];
        pos[idx3 + 1] = pos_vel[idx3 + 1];
        pos[idx3 + 2] = pos_vel[idx3 + 2];
        /* copy all masses to shared memory */
        extern __shared__ double mas[];
        mas[idx] = masses[idx];
        __syncthreads();
        /* copy [idx] position to register memory */
        double pos_idx[3] = {pos[idx3], pos[idx3 + 1], pos[idx3 + 2]};
        /* initialize [idx] acceleration on register memory */
        double acc[3] = {0.0, 0.0, 0.0};
        /* initialize coefs on register memory */
        double dist3, coef, diff[3];

        for (int j = 0; j < n; ++j) {
            diff[0] = pos[3 * j] - pos_idx[0];
            diff[1] = pos[3 * j + 1] - pos_idx[1];
            diff[2] = pos[3 * j + 2] - pos_idx[2];
            dist3 = cube(sqrt(sqr(diff[0]) + sqr(diff[1]) + sqr(diff[2])));
            coef = mas[j] / max(dist3, EPS);
            acc[0] += coef * diff[0];
            acc[1] += coef * diff[1];
            acc[2] += coef * diff[2];
        }
        result[n3 + idx3 + 0] = G * acc[0];
        result[n3 + idx3 + 1] = G * acc[1];
        result[n3 + idx3 + 2] = G * acc[2];
    }
}

void runge_kutta_2(const vector<double>& init, const vector<double>& masses, double t_max, double tau,
                   int print_every_n, const string& file_name) {
    ofstream stream(file_name);
    int n = masses.size();
    int size = init.size();  // 6 * n
    int step_num = 0;
    double t_cur = 0.0;
    vector<double> p_cur(init);
    stream << setprecision(10) << t_cur << " " << p_cur << '\n';

    double *d_p_cur, *d_p_mid, *d_masses, *d_k;
    hipMalloc(&d_p_cur, size * sizeof(double));
    hipMalloc(&d_p_mid, size * sizeof(double));
    hipMalloc(&d_masses, n * sizeof(double));
    hipMalloc(&d_k, size * sizeof(double));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsed_time;
    float exec_time = 0.0, copy_time = 0.0;

    hipEventRecord(start);
    /* copy masses and p(t) to device */
    hipMemcpy(d_p_cur, p_cur.data(), size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_masses, masses.data(), n * sizeof(double), hipMemcpyHostToDevice);
    /* end of record */
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    copy_time += elapsed_time;

    while (t_cur < t_max) {
        t_cur += tau;

        hipEventRecord(start);
        /* k(t) = { v(t), a(t) } */
        calc_k<<<(n + BS) / BS, BS>>>(d_k, d_p_cur, d_masses, n);
        /* p(t + tau/2) = p(t) + tau / 2 * k(t) */
        plus_product<<<(size + BS) / BS, BS>>>(d_p_mid, d_p_cur, tau / 2, d_k, size);
        /* k(t + tau/2) = { v(t + tau/2), a(t + tau/2) } */
        calc_k<<<(n + BS) / BS, BS>>>(d_k, d_p_mid, d_masses, n);
        /* p(t + tau) += tau * k(t + tau/2) */
        plus_equal_product<<<(size + BS) / BS, BS>>>(d_p_cur, tau, d_k, size);
        /* end of record */
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time, start, stop);
        exec_time += elapsed_time;

        hipEventRecord(start);
        /* copy p(t + tau) to host */
        hipMemcpy(p_cur.data(), d_p_cur, size * sizeof(double), hipMemcpyDeviceToHost);
        /* end of record */
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time, start, stop);
        copy_time += elapsed_time;

        if (++step_num % print_every_n == 0) {
            stream << setprecision(10) << t_cur << " " << p_cur << '\n';
        }
    }

    hipDeviceSynchronize();

    hipFree(d_p_cur);
    hipFree(d_p_mid);
    hipFree(d_masses);
    hipFree(d_k);
//    cout << "Time exec: " << exec_time / 1000.0 << endl;
//    cout << "Time copy: " << copy_time / 1000.0 << endl;
}

int main(int argc, char** argv) {
    // $./run N bs
    if (argc > 1) {
        N = stoi(argv[1]);
    }
    if (argc > 2) {
        BS = stoi(argv[2]);
    }
    cout << "N: " << N << "\nbs: " << BS << "\n";

    /* part 1 data init */
    vector<double> input_data = {1.0, 0.0, 0.0, 0.0, 0.9, 0.0,
                                 0.0, 1.0, 0.0, -0.9, 0.0, 0.0,
                                 -1.0, 0.0, 0.0, 0.0, -0.9, 0.0,
                                 0.0, -1.0, 0.0, 0.9, 0.0, 0.0};
    vector<double> masses = {8810324116.227, 8810324116.227, 8810324116.227, 8810324116.227};
    vector<double> init(input_data.size());
    for (int i = 0; i < init.size() / 6; ++i) {
        init[3 * i + 0] = input_data[6 * i + 0];
        init[3 * i + 1] = input_data[6 * i + 1];
        init[3 * i + 2] = input_data[6 * i + 2];
        init[init.size() / 2 + 3 * i + 0] = input_data[6 * i + 3];
        init[init.size() / 2 + 3 * i + 1] = input_data[6 * i + 4];
        init[init.size() / 2 + 3 * i + 2] = input_data[6 * i + 5];
    }

    /* part 1 calc */
    runge_kutta_2(init, masses, 20, 0.01, 10, "./cuda.txt");

    /* distributions init */
    std::random_device seeder;
    const auto seed = seeder.entropy() ? seeder() : time(nullptr);
    std::mt19937 rnd_gen(static_cast<std::mt19937::result_type>(seed));
    std::uniform_real_distribution<double> distr_pos(-10.0, 10.0);
    std::uniform_real_distribution<double> distr_vel(-1.0, 1.0);
    std::uniform_real_distribution<double> distr_mass(0.1, 10.0);
    auto rnd_pos = std::bind(distr_pos, rnd_gen);
    auto rnd_vel = std::bind(distr_vel, rnd_gen);
    auto rnd_mas = std::bind(distr_mass, rnd_gen);

    /* part 2 data init */
    init.resize(6 * N);
    for (int i = 0; i < 3 * N; ++i) {
        init[i] = rnd_pos();
    }
    for (int i = 3 * N; i < 6 * N; ++i) {
        init[i] = rnd_vel();
    }
    masses.resize(N);
    for (int i = 0; i < N; ++i) {
        masses[i] = rnd_mas();
    }

    /* part 2 calc */
    auto t1 = chrono::high_resolution_clock::now();
    runge_kutta_2(init, masses, 0.1, 0.05, 1, "./cuda_10000.txt");
    auto t2 = chrono::high_resolution_clock::now();
    cout << "Time cuda: " << chrono::duration_cast<chrono::milliseconds>(t2 - t1).count() / 1000.0 << endl;

    return 0;
}